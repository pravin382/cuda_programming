
#include <hip/hip_runtime.h>
#include <stdio.h>
#include "cublasXt.h"
#include <hiprand/hiprand.h>

void fill(double* &x, long m, long n, double val) {
  x = new double[m * n];
  for (long i = 0; i < m; ++i) {
    for (long j = 0; j < n; ++j) {
      x[i * n + j] = val;
    }
  }
}

int main() {
  cublasXtHandle_t xt_;
  
  if(cublasXtCreate(&xt_) != HIPBLAS_STATUS_SUCCESS) {
    printf("handle create fail\n"); 
    return 1;
  }
  int devices[1] = { 0 };  // add this line
  if(cublasXtDeviceSelect(xt_, 1, devices) != HIPBLAS_STATUS_SUCCESS) {
    printf("set devices fail\n");
    return 1;
  } // add this line


  double *A, *B, *C;
  long m = 10, n = 10, k = 20;

  fill(A, m, k, 0.2);
  fill(B, k, n, 0.3);
  fill(C, m, n, 0.0);

  double alpha = 1.0;
  double beta = 0.0;

  cublasXtDgemm(xt_, HIPBLAS_OP_N, HIPBLAS_OP_N,
    m, n, k, &alpha, A, m, B, k, &beta, C, m
  );

  hipDeviceSynchronize();

  for (int i = 0; i < m; ++i) {
    for (int j = 0; j < n; ++j) {
      printf ("%lf ", C[i *n + j]);
    }
    printf ("\n");
  }

  cublasXtDestroy(xt_);
  return 0;
}
